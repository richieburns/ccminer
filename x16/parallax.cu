#include "hip/hip_runtime.h"
/**
 * Parallax algorithm
 *
 * Nodescape 2018 - GPL code
 */

#include <stdio.h>
#include <memory.h>
#include <unistd.h>

extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"

#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x16.h"

static uint32_t *d_hash[MAX_GPUS];
// static __thread uint32_t s_ntime = UINT32_MAX;
static __thread int pViewPoint;
static __thread int pView[9]; 

static void getView(const uint32_t* prevblock, int *output)
{
	// Views:
	int nView[16][9] = {
		{ 0, 1, 2, 3, 4, 5, 6, 7, 8 },
		{ 0, 1, 2, 3, 5, 4, 6, 7, 8 },
		{ 0, 1, 2, 5, 3, 6, 4, 7, 8 },
		{ 0, 1, 5, 2, 6, 3, 7, 4, 8 },
		{ 0, 5, 1, 6, 2, 7, 3, 8, 4 },
		{ 5, 0, 6, 1, 7, 2, 8, 3, 4 },
		{ 5, 6, 0, 7, 1, 8, 2, 3, 4 },
		{ 5, 6, 7, 0, 8, 1, 2, 3, 4 },
		{ 5, 6, 7, 8, 0, 1, 2, 3, 4 },
		{ 5, 6, 7, 0, 8, 1, 2, 3, 4 },
		{ 5, 6, 0, 7, 1, 8, 2, 3, 4 },
		{ 5, 0, 6, 1, 7, 2, 8, 3, 4 },
		{ 0, 5, 1, 6, 2, 7, 3, 8, 4 },
		{ 0, 1, 5, 2, 6, 3, 7, 4, 8 },
		{ 0, 1, 2, 5, 3, 6, 4, 7, 8 },
		{ 0, 1, 2, 3, 5, 4, 6, 7, 8 }
	};

	uint8_t* data = (uint8_t*)prevblock;
	int sView = data[0] & 0xF;
	for (uint8_t i = 0; i < 9; i++) {
		output[i] = nView[sView][i];
	}
}

// Parallax CPU Hash (Validation)
extern "C" void parallax_hash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[128] = { 0 };

    static unsigned char pblank[1];

    sph_skein512_context     ctx_skein;     // 0
    sph_luffa512_context     ctx_luffa;     // 1
    sph_keccak512_context    ctx_keccak;    // 2
    sph_jh512_context        ctx_jh;        // 3
    sph_groestl512_context   ctx_groestl;   // 4
    sph_echo512_context      ctx_echo;      // 5
    sph_cubehash512_context  ctx_cubehash;  // 6
    sph_bmw512_context       ctx_bmw;       // 7
    sph_blake512_context     ctx_blake;     // 8      
    
    void *in = (void*) input;
	int size = 80;
	uint32_t *in32 = (uint32_t*) input;
    getView(&in32[1], pView);

	for (int i = 0; i < 9; ++i)
    {
		switch (pView[i]) {
			case 0:
				sph_skein512_init(&ctx_skein);
				sph_skein512(&ctx_skein, in, size);
				sph_skein512_close(&ctx_skein, hash);
				break;
			case 1:
				sph_luffa512_init(&ctx_luffa);
				sph_luffa512(&ctx_luffa, in, size);
				sph_luffa512_close(&ctx_luffa, hash);
				break;
			case 2:
				sph_keccak512_init(&ctx_keccak);
				sph_keccak512(&ctx_keccak, in, size);
				sph_keccak512_close(&ctx_keccak, hash);
				break;
			case 3:
				sph_jh512_init(&ctx_jh);
				sph_jh512(&ctx_jh, in, size);
				sph_jh512_close(&ctx_jh, hash);
				break;
			case 4:
				sph_groestl512_init(&ctx_groestl);
				sph_groestl512(&ctx_groestl, in, size);
				sph_groestl512_close(&ctx_groestl, hash);
				break;
			case 5:
				sph_echo512_init(&ctx_echo);
				sph_echo512(&ctx_echo, in, size);
				sph_echo512_close(&ctx_echo, hash);
				break;
			case 6:
				sph_cubehash512_init(&ctx_cubehash);
				sph_cubehash512(&ctx_cubehash, in, size);
				sph_cubehash512_close(&ctx_cubehash, hash);
				break;
			case 7:
				sph_bmw512_init(&ctx_bmw);
				sph_bmw512(&ctx_bmw, in, size);
				sph_bmw512_close(&ctx_bmw, hash);
				break;
			case 8:
				sph_blake512_init(&ctx_blake);
				sph_blake512(&ctx_blake, in, size);
				sph_blake512_close(&ctx_blake, hash);
				break;
		}
		in = (void*) hash;
		size = 64;
	}	
	memcpy(output, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

//#define _DEBUG
#define _DEBUG_PREFIX "parallax-"
#include "cuda_debug.cuh"

extern "C" int scanhash_parallax(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	const int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 20 : 19;
	if (strstr(device_name[dev_id], "GTX 1080")) intensity = 20;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		cuda_get_arch(thr_id);
		use_compat_kernels[thr_id] = (cuda_arch[dev_id] < 500);
		if (use_compat_kernels[thr_id])
			x11_echo512_cpu_init(thr_id, throughput);
		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_jh512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		qubit_luffa512_cpu_init(thr_id, throughput);
		x11_luffa512_cpu_init(thr_id, throughput); // 64
		x11_shavite512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput); // 64
		x16_echo512_cuda_init(thr_id, throughput);;

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	if (opt_benchmark) {
		((uint32_t*)ptarget)[7] = 0x003f;
		((uint8_t*)pdata)[8] = 0xAA; // hashOrder[0] = 'A'; for echo 80 + 64
	}

	uint32_t _ALIGN(64) endiandata[20];
	for (int k=0; k < 19; k++)
		be32enc(&endiandata[k], pdata[k]);
	// uint32_t ntime = swab32(pdata[17]);
	getView(&endiandata[1], pView);

	if (opt_debug && !thr_id) applog(LOG_DEBUG, "View %i, %i, %i, %i, %i, %i, %i, %i, %i", pView[0], pView[1], pView[2], pView[3], pView[4], pView[5], pView[6], pView[7], pView[8]);

	cuda_check_cpu_setTarget(ptarget);

	switch (pView[0]) {
		case 0:
			skein512_cpu_setBlock_80((void*)endiandata);
			break;
		case 5:
			x16_echo512_setBlock_80((void*)endiandata);
			break;
		default: {
			return -1;
		}
	}

	do {
		int order = 0;

		// Hash with CUDA

		switch (pView[0]) {
			case 0:
				skein512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], 1); order++;
				TRACE("skein80:");
				break;
			case 5:
				x16_echo512_cuda_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
				TRACE("echo   :");
				break;
		}

		for (int i = 1; i < 9; i++)
		{
			switch (pView[i]) {
				case 0:
					quark_skein512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					TRACE("skein  :");
					break;
				case 1:
					x11_luffa512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					TRACE("luffa  :");
					break;
				case 2:
					quark_keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					TRACE("keccak :");
					break;
				case 3:
					quark_jh512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					TRACE("jh512  :");
					break;
				case 4:
					quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					TRACE("groestl:");
					break;
				case 5:
					if (use_compat_kernels[thr_id])
						x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					else {
						x16_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); order++;
					}
					TRACE("echo   :");
					break;
				case 6:
					x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					TRACE("cube   :");
					break;
				case 7:
					quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					TRACE("bmw    :");
					break;
				case 8:
					quark_blake512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
					TRACE("blake  :");
					break;
			}
		}

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);

#ifdef _DEBUG
		uint32_t _ALIGN(64) dhash[8];
		be32enc(&endiandata[19], pdata[19]);
		parallax_hash(dhash, endiandata);
		applog_hash(dhash);
		return -1;
#endif
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			parallax_hash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					parallax_hash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}

				return work->valid_nonces;
			} else if (vhash[7] > Htarg) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t)throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_parallax(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);

	hipDeviceSynchronize();
	init[thr_id] = false;
}
